#include "hip/hip_runtime.h"
#include <tensorProc.cuh>

//template<typename T>
__global__ void preprocessKernel(float *input_image,  float *bgrChannels_0,
        float *bgrChannels_1, float *bgrChannels_2, int input_height, int input_width) {
    printf("test hhhhh\n");
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
#if 0
    printf("test hh\n");
    if (idx < input_width && idy < input_height) {
        int index = idy * input_width + idx;
        bgrChannels_0[index] = (bgrChannels_0[index] / 128.0) - (127.5 / 128.0);
        bgrChannels_1[index] = (bgrChannels_1[index] / 128.0) - (127.5 / 128.0);
        bgrChannels_2[index] = (bgrChannels_2[index] / 128.0) - (127.5 / 128.0);
        input_image[index] = bgrChannels_0[index];
        input_image[index + input_height * input_width] = bgrChannels_1[index];
        input_image[index + 2 * input_height * input_width] = bgrChannels_2[index];
    }
#endif
}

void  preprocessKernelFloat(float *input_image,  float *bgrChannels_0,
    float *bgrChannels_1, float *bgrChannels_2, int input_height, int input_width) {
    dim3 blockSize(16, 16);
    dim3 gridSize((input_width + blockSize.x - 1) / blockSize.x,
            (input_height + blockSize.y - 1) / blockSize.y);

    printf("test hahaha");
    preprocessKernel<<<gridSize, blockSize>>>(input_image, bgrChannels_0,
            bgrChannels_1, bgrChannels_2, input_height, input_width);

    hipDeviceSynchronize();
}
